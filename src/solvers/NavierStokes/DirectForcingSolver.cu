#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file DirectForcingSolver.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the methods of the class \c DirectForcingSolver.
 */


#include "DirectForcingSolver.h"
#include <sys/stat.h>
#include <thrust/extrema.h>
#include <cusp/io/matrix_market.h>

/**
 * \brief Constructor. Initializes the simulation parameters and the domain info.
 */
template <typename memoryType>
DirectForcingSolver<memoryType>::DirectForcingSolver(parameterDB *pDB, domain *dInfo)
{
	NavierStokesSolver<memoryType>::paramDB = pDB;
	NavierStokesSolver<memoryType>::domInfo = dInfo;
}

/**
 * \brief Initialize the vectors used in the simulation.
 */
template <typename memoryType>
void DirectForcingSolver<memoryType>::initialise()
{
	int nx = NavierStokesSolver<memoryType>::domInfo->nx,
        ny = NavierStokesSolver<memoryType>::domInfo->ny;

	int numUV = (nx-1)*ny + nx*(ny-1);
	int numP  = nx*ny;
	
	NavierStokesSolver<memoryType>::initialiseCommon();
	
	NSWithBody<memoryType>::initialiseBodies();
	
	NavierStokesSolver<memoryType>::initialiseArrays(numUV, numP);
	
	NavierStokesSolver<memoryType>::logger.startTimer("allocateMemory");

	tags.resize(numUV);
	tagsD.resize(numUV);
	tags2.resize(numUV);
	tags2D.resize(numUV);
	coeffs.resize(numUV);
	coeffsD.resize(numUV);
	coeffs2.resize(numUV);
	coeffs2D.resize(numUV);
	uv.resize(numUV);
	uvD.resize(numUV);

	pressure.resize(numP);
	cusp::blas::fill(pressure, 0.0);

	NavierStokesSolver<memoryType>::logger.startTimer("allocateMemory");
	
	tagPoints();
	std::cout << "Done tagging points!" << std::endl;
	
	NavierStokesSolver<memoryType>::assembleMatrices();
}

/**
 * \brief Updates the matrices every time the body is moved.
 *
 * Change the location of the body points, re-tags all the points on the 
 * velocity grid to locate the new forcing nodes. Reassembles the 
 * matrices. Moving bodies have not been tested for DirectForcingSolver.
 */
template <typename memoryType>
void DirectForcingSolver<memoryType>::updateSolverState()
{
	if (NSWithBody<memoryType>::B.bodiesMove)
	{
		// update the locations of the body points
		NSWithBody<memoryType>::updateBodies();
		
		// retag points
		tagPoints();
		
		// assemble the matrices generated using new tags
		NavierStokesSolver<memoryType>::assembleMatrices();
	}
}

/**
 * \brief Assembles the matrix rhs1 for DirectForcingSolver.
 *
 * This function first calls the function assembleRHS1 from NavierStokesSolver.
 * Then it called the function updateRHS1 to modify only the elements of the 
 * vector that correspond to the interpolation nodes.
 */
template <typename memoryType>
void DirectForcingSolver<memoryType>::assembleRHS1()
{
	NavierStokesSolver<memoryType>::assembleRHS1();
	
	NavierStokesSolver<memoryType>::logger.startTimer("updateRHS1");
	updateRHS1();
	NavierStokesSolver<memoryType>::logger.startTimer("updateRHS1");
}

/**
 * \brief Prints the min, max and sum of the divergences of the velocity field 
 *        in every cell of the domain.
 *
 * The divergence is calculated as QT*q, which is technically the sum of the 
 * mass fluxes in every cell. This QT is also differs depending on the Direct 
 * Forcing method used.
 */
template <typename memoryType>
void DirectForcingSolver<memoryType>::writeMassFluxInfo()
{
	parameterDB  &db = *NavierStokesSolver<memoryType>::paramDB;
	int     nx = NavierStokesSolver<memoryType>::domInfo->nx,
	        ny = NavierStokesSolver<memoryType>::domInfo->ny,
	        timeStep = NavierStokesSolver<memoryType>::timeStep;

	cusp::array1d<real, memoryType> fluxes(nx*ny);
	cusp::multiply(NavierStokesSolver<memoryType>::QT, NavierStokesSolver<memoryType>::q, fluxes);
	int minPosition = thrust::min_element(fluxes.begin(), fluxes.end()) - fluxes.begin(),
	    maxPosition = thrust::max_element(fluxes.begin(), fluxes.end()) - fluxes.begin();
	real minFlux = fluxes[minPosition],
	     maxFlux = fluxes[maxPosition],
	     globalSum = thrust::reduce(fluxes.begin(), fluxes.end());

	std::ofstream fluxInfoFile;
	std::string folder = db["inputs"]["caseFolder"].get<std::string>();
	std::stringstream out;
	out << folder << "/massFlux";
	
	if(timeStep==1)
		fluxInfoFile.open(out.str().c_str());
	else
		fluxInfoFile.open(out.str().c_str(), std::ios::out | std::ios::app);
		
	fluxInfoFile << timeStep << '\t' << minFlux << '\t' << maxFlux << '\t' << globalSum << std::endl;
	fluxInfoFile.close();
}

/**
 * \brief Projects the pressure gradient on to the intermediate velocity field
 *        to obtain the divergence-free velocity field at the next time step.
 */
template <typename memoryType>
void DirectForcingSolver<memoryType>::projectionStep()
{
	NavierStokesSolver<memoryType>::projectionStep();

	NavierStokesSolver<memoryType>::logger.startTimer("projectionStep");
	cusp::blas::axpy(NavierStokesSolver<memoryType>::lambda, pressure , 1.0);
	NavierStokesSolver<memoryType>::logger.stopTimer("projectionStep");
}

/**
 * \brief Writes the velocity, pressure, force and mass flux data at every save point.
 */
template <typename memoryType>
void DirectForcingSolver<memoryType>::writeData()
{	
	NavierStokesSolver<memoryType>::logger.startTimer("output");

	parameterDB  &db = *NavierStokesSolver<memoryType>::paramDB;
	real         dt  = db["simulation"]["dt"].get<real>();
	int          timeStep = NavierStokesSolver<memoryType>::timeStep;

	NSWithBody<memoryType>::writeCommon();
	
	// Print forces calculated using the CV approach
	NSWithBody<memoryType>::calculateForce();
	NSWithBody<memoryType>::forceFile << timeStep*dt << '\t' << NSWithBody<memoryType>::forceX << '\t' << NSWithBody<memoryType>::forceY << std::endl;

	writeMassFluxInfo();
	
	NavierStokesSolver<memoryType>::logger.stopTimer("output");
}

/**
 * \brief Generates the right-hand side matrix in the Poisson step.
 *
 * Because the fully discrete direct forcing method separates the domains 
 * inside and outside the immersed boundary, and Neumann boundary conditions
 * are enforced at the immersed boundary, a point also needs to be fixed 
 * inside the immersed boundary, so that Poisson system does not have any 
 * zero eigenvalues.
 *
 * In this function, \a phi at a point that corresponds to the center of the 
 * grid is fixed as zero. This is the cell with indices (nx/2, ny/2), and is 
 * the center in terms of the indices and not the physical location in space.
 * Of course, this is invalid if the interior of the body does not include 
 * this point, and the solution obtained will be unphysical.
 *
 * This needs to be done in a better way (i.e. by locating points that 
 * are inside the immersed boundaries, and fixing them to zero)
 */
template <typename memoryType>
void DirectForcingSolver<memoryType>::generateC()
{
	int nx = NavierStokesSolver<memoryType>::domInfo->nx,
	    ny = NavierStokesSolver<memoryType>::domInfo->ny;
	int index = 5*(ny/2)*nx - nx - ny + 5*(nx/2) - 1 + 2;
	int row = (ny/2)*nx+nx/2;

	NavierStokesSolver<memoryType>::generateC();
	bool flag = true;
	while(flag)
	{
		if(NavierStokesSolver<memoryType>::C.row_indices[index]==NavierStokesSolver<memoryType>::C.column_indices[index] && NavierStokesSolver<memoryType>::C.column_indices[index]==row)
		{
			NavierStokesSolver<memoryType>::C.values[index] += NavierStokesSolver<memoryType>::C.values[index];
			flag = false;
		}
		index++;
	}
}

// inline files in the folder "DirectForcing"
#include "DirectForcing/tagPoints.inl"
#include "DirectForcing/generateL.inl"
#include "DirectForcing/generateA.inl"
#include "DirectForcing/updateRHS1.inl"
#include "DirectForcing/generateQT.inl"

// specialization of the class
template class DirectForcingSolver<host_memory>;
template class DirectForcingSolver<device_memory>;
