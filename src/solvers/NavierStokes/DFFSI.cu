#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file DFFSI.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the methods of the class \c DFFSI.
 */


#include "DFFSI.h"
#include <sys/stat.h>
#include <thrust/extrema.h>
#include <cusp/io/matrix_market.h>
#include <ctime>

/**
 * \brief Constructor. Initializes the simulation parameters and the domain info.
 */
template <typename memoryType>
DFFSI<memoryType>::DFFSI(parameterDB *pDB, domain *dInfo)
{
	NavierStokesSolver<memoryType>::paramDB = pDB;
	NavierStokesSolver<memoryType>::domInfo = dInfo;
}

/**
 * \brief Initialize the vectors used in the simulation.
 */
template <typename memoryType>
void DFFSI<memoryType>::initialise()
{
	int nx = NavierStokesSolver<memoryType>::domInfo->nx,
        ny = NavierStokesSolver<memoryType>::domInfo->ny;

	int numUV = (nx-1)*ny + nx*(ny-1);
	int numP  = nx*ny;
	
	NavierStokesSolver<memoryType>::initialiseCommon();
	
	NSWithBody<memoryType>::initialiseBodies();
	
	NavierStokesSolver<memoryType>::initialiseArrays(numUV, numP);
	
	NavierStokesSolver<memoryType>::logger.startTimer("allocateMemory");

	tags.resize(numUV);
	tagsD.resize(numUV);
	tags2.resize(numUV);
	tags2D.resize(numUV);
	coeffs.resize(numUV);
	coeffsD.resize(numUV);
	coeffs2.resize(numUV);
	coeffs2D.resize(numUV);
	uv.resize(numUV);
	uvD.resize(numUV);
	q_is.resize(numUV);
	A_is.resize(numUV);
	lambda_is.resize(numUV);
	tempForce.resize(numUV);

	pressure.resize(numP);
	cusp::blas::fill(pressure, 0.0);

	NavierStokesSolver<memoryType>::logger.startTimer("allocateMemory");
	
	tagPoints();//tagged for update
	std::cout << "Done tagging points!" << std::endl;
	
	NavierStokesSolver<memoryType>::assembleMatrices();
}

template <typename memoryType>
void DFFSI<memoryType>::stepTime()
{
	NavierStokesSolver<memoryType>::qOld = NavierStokesSolver<memoryType>::q;
	NSWithBody<memoryType>::B.yk = NSWithBody<memoryType>::B.y;
	//NSWithBody<memoryType>::B.xk = NSWithBody<memoryType>::B.x;
	NSWithBody<memoryType>::B.vBk = NSWithBody<memoryType>::B.vB;
	//NSWithBody<memoryType>::B.uBk = NSWithBody<memoryType>::B.uB;
	int k = 1; 

	if (NavierStokesSolver<memoryType>::timeStep < 1) //no fsi
	{
		DFFSI<memoryType>::updateSolverState();
		NavierStokesSolver<memoryType>::generateRN();
		NavierStokesSolver<memoryType>::generateBC1();
		DFFSI<memoryType>::assembleRHS1();
		NavierStokesSolver<memoryType>::solveIntermediateVelocity();

		// Set up and solve the Poisson system
		NavierStokesSolver<memoryType>::generateBC2();
		NavierStokesSolver<memoryType>::assembleRHS2();
		NavierStokesSolver<memoryType>::solvePoisson();

		// Projection step
		DFFSI<memoryType>::projectionStep(); //tagged for update

		DFFSI<memoryType>::calculateForce();
		printFSI();
	}
	else //fsi
	{
		do
		{
		//updates required for a moving body
		DFFSI<memoryType>::updateSolverState(); //qt q c pc2

		//intermediate velocity
		NavierStokesSolver<memoryType>::generateRN();
		NavierStokesSolver<memoryType>::generateBC1();
		DFFSI<memoryType>::assembleRHS1();
		NavierStokesSolver<memoryType>::solveIntermediateVelocity();

		// Set up and solve the Poisson system
		NavierStokesSolver<memoryType>::generateBC2();
		NavierStokesSolver<memoryType>::assembleRHS2();
		NavierStokesSolver<memoryType>::solvePoisson();

		// Projection step
		DFFSI<memoryType>::projectionStep();

		//FSI
		DFFSI<memoryType>::calculateForce();
		NSWithBody<memoryType>::B.converged[0] = true;
		DFFSI<memoryType>::solveStructure();

		k++;
		}while(k<1);
		//}while (NSWithBody<memoryType>::B.converged[0] == false);
	}
	NavierStokesSolver<memoryType>::timeStep++;
}

/**
 * \brief Updates the matrices every time the body is moved.
 *
 * Change the location of the body points, re-tags all the points on the
 * velocity grid to locate the new forcing nodes. Reassembles the
 * matrices. Moving bodies have not been tested for DFFSI.
 */
template <typename memoryType>
void DFFSI<memoryType>::updateSolverState()
{
	//if (fabs(NSWithBody<memoryType>::B.y[0]) > 0.001)
	{
	//calculateCellIndicies
	NavierStokesSolver<memoryType>::logger.startTimer("updateSolverState");
	NSWithBody<memoryType>::B.calculateCellIndices(*NavierStokesSolver<memoryType>::domInfo);
	NSWithBody<memoryType>::B.calculateBoundingBoxes(*NavierStokesSolver<memoryType>::paramDB,  *NavierStokesSolver<memoryType>::domInfo);
	// tag points
	NavierStokesSolver<memoryType>::logger.startTimer("updateSolverState_tagPoints");
	tagPoints();
	NavierStokesSolver<memoryType>::logger.stopTimer("updateSolverState_tagPoints");
	
	//assembleMatrices
	//generateL
	generateL();
	//generateA
	generateA(NavierStokesSolver<memoryType>::intgSchm.alphaImplicit[NavierStokesSolver<memoryType>::subStep]);
	//update PC1
	NavierStokesSolver<memoryType>::PC1 ->update(NavierStokesSolver<memoryType>::A);
	//generateQT
	generateQT();
	//generateC
	NavierStokesSolver<memoryType>::logger.startTimer("updateSolverState_generateC");
	generateC(); // QT*BN*Q
	NavierStokesSolver<memoryType>::logger.stopTimer("updateSolverState_generateC");
	//Update PC2
	NavierStokesSolver<memoryType>::PC2 ->update(NavierStokesSolver<memoryType>::C);

	NavierStokesSolver<memoryType>::logger.stopTimer("updateSolverState");
	}
}

template<typename memoryType>
void DFFSI<memoryType>::printFSI()
{
	std::cout<<NavierStokesSolver<memoryType>::timeStep*0.01<<"\t"<<NSWithBody<memoryType>::B.ykp1[0]<<"\t"<<NSWithBody<memoryType>::B.forceY[0]<<"\n";
	NSWithBody<memoryType>::output<<NavierStokesSolver<memoryType>::timeStep*0.01<<"  \t";
	NSWithBody<memoryType>::output<<NSWithBody<memoryType>::B.ykp1[0]<<"  \t";
	NSWithBody<memoryType>::output<<NSWithBody<memoryType>::B.y[0]<<"  \t";
	NSWithBody<memoryType>::output<<NSWithBody<memoryType>::B.yk[0]<<"  \t";
	NSWithBody<memoryType>::output<<NSWithBody<memoryType>::B.forceY[0]<<"  \t";
	NSWithBody<memoryType>::output<<NSWithBody<memoryType>::B.forceX[0]<<"  \t";
	NSWithBody<memoryType>::output<<NSWithBody<memoryType>::B.vBk[0]<<"   \t";
	NSWithBody<memoryType>::output<<NSWithBody<memoryType>::B.vB[0]<<"   \n";
}

/**
 * \brief Assembles the matrix rhs1 for DFFSI.
 *
 * This function first calls the function assembleRHS1 from NavierStokesSolver.
 * Then it called the function updateRHS1 to modify only the elements of the 
 * vector that correspond to the interpolation nodes.
 */
template <typename memoryType>
void DFFSI<memoryType>::assembleRHS1()
{
	NavierStokesSolver<memoryType>::assembleRHS1();
	
	NavierStokesSolver<memoryType>::logger.startTimer("updateRHS1");
	updateRHS1();
	NavierStokesSolver<memoryType>::logger.startTimer("updateRHS1");
}

/**
 * \brief Prints the min, max and sum of the divergences of the velocity field 
 *        in every cell of the domain.
 *
 * The divergence is calculated as QT*q, which is technically the sum of the 
 * mass fluxes in every cell. This QT is also differs depending on the Direct 
 * Forcing method used.
 */
template <typename memoryType>
void DFFSI<memoryType>::writeMassFluxInfo()
{
	parameterDB  &db = *NavierStokesSolver<memoryType>::paramDB;
	int     nx = NavierStokesSolver<memoryType>::domInfo->nx,
	        ny = NavierStokesSolver<memoryType>::domInfo->ny,
	        timeStep = NavierStokesSolver<memoryType>::timeStep;

	cusp::array1d<real, memoryType> fluxes(nx*ny);
	cusp::multiply(NavierStokesSolver<memoryType>::QT, NavierStokesSolver<memoryType>::q, fluxes);
	int minPosition = thrust::min_element(fluxes.begin(), fluxes.end()) - fluxes.begin(),
	    maxPosition = thrust::max_element(fluxes.begin(), fluxes.end()) - fluxes.begin();
	real minFlux = fluxes[minPosition],
	     maxFlux = fluxes[maxPosition],
	     globalSum = thrust::reduce(fluxes.begin(), fluxes.end());

	std::ofstream fluxInfoFile;
	std::string folder = db["inputs"]["caseFolder"].get<std::string>();
	std::stringstream out;
	out << folder << "/massFlux";
	
	if(timeStep==1)
		fluxInfoFile.open(out.str().c_str());
	else
		fluxInfoFile.open(out.str().c_str(), std::ios::out | std::ios::app);
		
	fluxInfoFile << timeStep << '\t' << minFlux << '\t' << maxFlux << '\t' << globalSum << std::endl;
	fluxInfoFile.close();
}

/**
 * \brief Projects the pressure gradient on to the intermediate velocity field
 *        to obtain the divergence-free velocity field at the next time step.
 */
template <typename memoryType>
void DFFSI<memoryType>::projectionStep()
{
	NavierStokesSolver<memoryType>::projectionStep();

	NavierStokesSolver<memoryType>::logger.startTimer("projectionStep");
	cusp::blas::axpy(NavierStokesSolver<memoryType>::lambda, pressure , 1.0);
	NavierStokesSolver<memoryType>::logger.stopTimer("projectionStep");
}

/**
 * \brief Writes the velocity, pressure, force and mass flux data at every save point.
 */
template <typename memoryType>
void DFFSI<memoryType>::writeData()
{	
	NavierStokesSolver<memoryType>::logger.startTimer("output");

	parameterDB  &db = *NavierStokesSolver<memoryType>::paramDB;
	real         dt  = db["simulation"]["dt"].get<real>();
	int          timeStep = NavierStokesSolver<memoryType>::timeStep;

	NSWithBody<memoryType>::writeCommon();
	
	// Print forces calculated using the CV approach
	//NSWithBody<memoryType>::calculateForce();
	NSWithBody<memoryType>::forceFile << timeStep*dt << '\t' << NSWithBody<memoryType>::forceX << '\t' << NSWithBody<memoryType>::forceY << std::endl;

	writeMassFluxInfo();
	
	NavierStokesSolver<memoryType>::logger.stopTimer("output");
}

/**
 * \brief Generates the right-hand side matrix in the Poisson step.
 *
 * Because the fully discrete direct forcing method separates the domains 
 * inside and outside the immersed boundary, and Neumann boundary conditions
 * are enforced at the immersed boundary, a point also needs to be fixed 
 * inside the immersed boundary, so that Poisson system does not have any 
 * zero eigenvalues.
 *
 * In this function, \a phi at a point that corresponds to the center of the 
 * grid is fixed as zero. This is the cell with indices (nx/2, ny/2), and is 
 * the center in terms of the indices and not the physical location in space.
 * Of course, this is invalid if the interior of the body does not include 
 * this point, and the solution obtained will be unphysical.
 *
 * This needs to be done in a better way (i.e. by locating points that 
 * are inside the immersed boundaries, and fixing them to zero)
 */
template <typename memoryType>
void DFFSI<memoryType>::generateC()
{
	int nx = NavierStokesSolver<memoryType>::domInfo->nx,
	    ny = NavierStokesSolver<memoryType>::domInfo->ny;
	int index = 5*(ny/2)*nx - nx - ny + 5*(nx/2) - 1 + 2;
	int row = (ny/2)*nx+nx/2;

	NavierStokesSolver<memoryType>::generateC();
	bool flag = true;
	while(flag)
	{
		if(NavierStokesSolver<memoryType>::C.row_indices[index]==NavierStokesSolver<memoryType>::C.column_indices[index] && NavierStokesSolver<memoryType>::C.column_indices[index]==row)
		{
			NavierStokesSolver<memoryType>::C.values[index] += NavierStokesSolver<memoryType>::C.values[index];
			flag = false;
		}
		index++;
	}
}

// inline files in the folder "DFFSI"
#include "DFFSI/tagPoints.inl"
#include "DFFSI/generateL.inl"
#include "DFFSI/generateA.inl"
#include "DFFSI/updateRHS1.inl"
#include "DFFSI/generateQT.inl"
#include "DFFSI/calculateForce.inl"
#include "DFFSI/solveStructure.inl"

// specialization of the class
template class DFFSI<host_memory>;
template class DFFSI<device_memory>;
