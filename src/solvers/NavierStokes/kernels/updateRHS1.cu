#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file updateRHS1.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the kernels to update the right hand-side
 *        of the intermediate velocity flux solver.
 *        It replaces the right-hand side elements at the tagged points, with
 *        values obtained from the interpolation relations at those points.
 */


#include "updateRHS1.h"


#define BSZ 16


/**
 * \namespace kernels
 * \brief Contains all custom-written CUDA kernels.
 */
namespace kernels
{

// 1-d interpolation
/**
 * \brief To be documented.
 */
__global__
void updateRHS1(real *rhs1, int numUV, int *tags)
{
	int	I = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(I>=numUV)
		return;
	
	rhs1[I] = rhs1[I]*(tags[I]==-1);
}

/**
 * \brief To be documented.
 */
__global__
void updateRHS1X(real *rhs1, int nx, int ny, real dt, real *dx, int *tags, real *coeffs, real *uv)
{
	int	I = blockIdx.x*blockDim.x + threadIdx.x;
	int i = I % (nx-1);
	
	if( I < (nx-1)*ny )
	{
		rhs1[I] = (tags[I]==-1)*rhs1[I] 
		          + ((tags[I]!=-1)*((1.0-coeffs[I])*uv[I])) * 0.5*(dx[i+1]+dx[i])/dt;
	}
}

/**
 * \brief To be documented.
 */
__global__
void updateRHS1Y(real *rhs1, int nx, int ny, real dt, real *dy, int *tags, real *coeffs, real *uv)
{
	int numU = (nx-1)*ny;
	int	I = blockIdx.x*blockDim.x + threadIdx.x + numU;
	int j = (I-numU) / nx;
	
	if( I < numU + nx*(ny-1) )
	{
		rhs1[I] = (tags[I]==-1)*rhs1[I] 
		          + ((tags[I]!=-1)*((1.0-coeffs[I])*uv[I])) * 0.5*(dy[j+1]+dy[j])/dt;
	}
}

__global__
void updateRHS1X(real *rhs1, int nx, int ny, real dt, real *dx, int *tags, real *coeffs, real *coeffs2, real *uv)
{
	int	I = blockIdx.x*blockDim.x + threadIdx.x;
	int i = I % (nx-1);
	
	if( I < (nx-1)*ny )
	{
		rhs1[I] = (tags[I]==-1)*rhs1[I] 
		          + ((tags[I]!=-1)*((1.0-coeffs[I]-coeffs2[I])*uv[I])) * 0.5*(dx[i+1]+dx[i])/dt;
	}
}

__global__
void updateRHS1Y(real *rhs1, int nx, int ny, real dt, real *dy, int *tags, real *coeffs, real *coeffs2, real *uv)
{
	int numU = (nx-1)*ny;
	int	I = blockIdx.x*blockDim.x + threadIdx.x + numU;
	int j = (I-numU) / nx;
	
	if( I < numU + nx*(ny-1) )
	{
		rhs1[I] = (tags[I]==-1)*rhs1[I] 
		          + ((tags[I]!=-1)*((1.0-coeffs[I]-coeffs2[I])*uv[I])) * 0.5*(dy[j+1]+dy[j])/dt;
	}
}

} // end of namespace kernels
