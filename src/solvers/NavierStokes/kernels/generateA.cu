#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file generateA.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the kernels required to generate the matrix
 *        resulting from the implicit terms in the momentum equation.
 */


#include "generateA.h"


/**
 * namespace kernels
 * \brief Contains all the custom-written CUDA kernels.
 */
namespace kernels
{

/**
 * \brief Generates a block of the matrix resulting from implicit terms in the momentum equation.
 *
 * It assembles the matrix \c A as a combination 
 * of the Laplacian matrix \c L and the mass matrix \c M.
 * A = M-alpha*L
 * The parameter alpha is the coefficient of the implicit part of the diffusion term.
 * It is 1 for a backward Euler scheme, 0.5 for a Crank-Nicolson scheme, and 0 for a fully explicit scheme.
 *
 * \param ARows rows of the COO matrix \c A
 * \param ACols columns of the COO matrix \c A
 * \param AVals values of the COO matrix \c A
 * \param MVals values of the COO matrix \c M
 * \param LRows rows of the COO matrix \c L
 * \param LCols columns of the COO matrix \c L
 * \param LVals values of the COO matrix \c A
 * \param ASize number of entries of the COO matrix \c A
 * \param alpha implicit coefficient of the diffusive scheme
 */
__global__
void generateA(int *ARows, int *ACols, real *AVals, real *MVals, int *LRows, int *LCols, real *LVals, int ASize, real alpha)
{
	for (int I=threadIdx.x + blockIdx.x*blockDim.x; I<ASize; I += blockDim.x*gridDim.x)
	{
		ARows[I] = LRows[I];
		ACols[I] = LCols[I];
		AVals[I] = -alpha*LVals[I] + (LRows[I]==LCols[I])*MVals[LRows[I]];
	}
}

/**
 * \brief Generates a block of the matrix resulting from implicit terms in the momentum equation
 *        for the direct forcing method.
 *
 * It assembles the matrix \c A as a combination 
 * of the Laplacian matrix \c L and the mass matrix \c M.
 * The parameter alpha is the coefficient of the implicit part of the diffusion term.
 * It is 1 for a backward Euler scheme, 0.5 for a Crank-Nicolson scheme, and 0 for a fully explicit scheme.
 * The left-hand side matrix A is set up as M-alpha*L, where M is the mass matrix, and L the Laplacian matrix.
 * But in the case of the direct forcing method, some rows are determined by interpolation relations,
 * and the rows of L are modified appropriately. For these rows alone, the rows of A are given by M-L.
 *
 * \param ARows rows of the COO matrix \c A
 * \param ACols columns of the COO matrix \c A
 * \param AVals values of the COO matrix \c A
 * \param MVals values of the COO matrix \c M
 * \param LRows rows of the COO matrix \c L
 * \param LCols columns of the COO matrix \c L
 * \param LVals values of the COO matrix \c A
 * \param ASize number of entries of the COO matrix \c A
 * \param alpha implicit coefficient of the diffusive scheme
 * \param tagsX tag to check if the node is next to an immersed boundary
 * \param tagsY tag to check if the node is next to an immersed boundary
 */
__global__
void generateADirectForcing(int *ARows, int *ACols, real *AVals, real *MVals, int *LRows, int *LCols, real *LVals, int ASize, real alpha, int *tags)
{
	for(int I=threadIdx.x + blockIdx.x*blockDim.x; I<ASize; I += blockDim.x*gridDim.x)
	{
		ARows[I] = LRows[I];
		ACols[I] = LCols[I];
		AVals[I] =   (tags[LRows[I]] == -1)*(-alpha*LVals[I]) // if the current location is untagged, add -alpha*L
		           + (tags[LRows[I]] != -1)*(-LVals[I]) // if the current location is tagged, add -L
		           + (LRows[I]==LCols[I])*MVals[LRows[I]]; // if it is a diagonal, add M
	}
}
	
} // end of namespace kernels
