#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file generateM.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the kernels to generate the mass matrix and its inverse.
 */


#include "generateM.h"


/**
 * \namespace kernels
 * \brief Contains all custom-written CUDA kernels.
 */
namespace kernels
{

/**
 * \brief Computes an element of the mass matrix and its inverse for a x-velocity node.
 *
 * \param MRows row index of elements of the mass matrix
 * \param MCols column index of elements of the mass matrix
 * \param MVals value of elements of the mass matrix
 * \param MinvRows row index of elements of the mass matrix inverse
 * \param MinvCols column index of elements of the mass matrix inverse
 * \param MinvVals value of elements of the mass matrix inverse
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 * \param dx cell-widths in the x-direction
 * \param dy cell-widths in the y-direction
 * \param dt time-increment
 */
__global__
void fillM_u(int *MRows, int *MCols, real *MVals, int *MinvRows, int *MinvCols, real *MinvVals, int nx, int ny, real *dx, real *dy, real dt)
{
	int I = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(I >= (nx-1)*ny) return;
	
	int  i = I % (nx-1);
	int  j = I / (nx-1);
	real value = 0.5*(dx[i]+dx[i+1])/dy[j]/dt;
	
	MRows[I] = I;
	MCols[I] = I;
	MVals[I] = value;
	
	MinvRows[I] = I;
	MinvCols[I] = I;
	MinvVals[I] = 1.0/value;
}

/**
 * \brief Computes an element of the mass matrix and its inverse for a y-velocity node.
 *
 * \param MRows row index of elements of the mass matrix
 * \param MCols column index of elements of the mass matrix
 * \param MVals value of elements of the mass matrix
 * \param MinvRows row index of elements of the mass matrix inverse
 * \param MinvCols column index of elements of the mass matrix inverse
 * \param MinvVals value of elements of the mass matrix inverse
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 * \param dx cell-widths in the x-direction
 * \param dy cell-widths in the y-direction
 * \param dt time-increment
 */
__global__
void fillM_v(int *MRows, int *MCols, real *MVals, int *MinvRows, int *MinvCols, real *MinvVals, int nx, int ny, real *dx, real *dy, real dt)
{
	int I = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(I >= nx*(ny-1)) return;
	
	int  numU = (nx-1)*ny;
	int  i = I % nx;
	int  j = I / nx;
	real value = 0.5*(dy[j]+dy[j+1])/dx[i]/dt;
	
	MRows[I+numU] = I+numU;
	MCols[I+numU] = I+numU;
	MVals[I+numU] = value;
	
	MinvRows[I+numU] = I+numU;
	MinvCols[I+numU] = I+numU;
	MinvVals[I+numU] = 1.0/value;
}

} // end of namespace kernels
