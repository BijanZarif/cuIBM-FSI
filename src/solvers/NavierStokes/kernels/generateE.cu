#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file generateE.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the kernels to generate elements of the interpolation matrix.
 */


#include "generateE.h"


/**
 * \brief Discrete delta function defined by Roma et al. (1999).
 *
 * \param x x- or y- component of the vector defined between two points
 * \param h the grid-spacing
 *
 * \return the value of the discrete delta function
 */
__device__ \
real dhRomaDeviceE(real x, real h)
{
	real r = fabs(x)/h;
	
	if(r>1.5)
		return 0.0;
	else if(r>0.5 && r<=1.5)
		return 1.0/(6*h)*( 5.0 - 3.0*r - sqrt(-3.0*(1-r)*(1-r) + 1.0) );
	else
		return 1.0/(3*h)*( 1.0 + sqrt(-3.0*r*r + 1.0) );
}

/**
 * \brief Two-dimension discrete delta function.
 *
 * \param x x-component of the vector defined between two points
 * \param y y-component of the vector defined between two points
 * \param h the grid-spacing
 *
 * \return the value of the discrete delta function in 2D
 */
__device__ \
real deltaDeviceE(real x, real y, real h)
{
	return dhRomaDeviceE(x, h) * dhRomaDeviceE(y, h);
}

/**
 * \namespace kernels
 * \brief Contains all custom-written CUDA kernels.
 */
namespace kernels
{
	
/**
 * \brief Generates the interpolation matrix (on the host).
 *
 * \param ERows row index of elements of the interpolation matrix
 * \param ECols column index of elements of the interpolation matrix
 * \param EVals value of elements of the interpolation matrix
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 * \param x x-component of grid points
 * \param y y-component of grid points
 * \param dx cell-widths in the x-direction
 * \param totalPoints number of body points (all bodies included)
 * \param xB x-coordinate of body points (all bodies included)
 * \param yB y-coordinate of body points (all bodies included)
 * \param I x-index of the cells in which body points are located
 * \param J y-index of the cells in which body points are located
 */
void generateEHost(int *ERows,  int *ECols,  real *EVals,
                   int nx, int ny, real *x, real *y, real *dx,
                   int totalPoints, real *xB, real *yB, int *I, int *J)
{
	for(int bodyIdx=0; bodyIdx<totalPoints; bodyIdx++)
	{
		int  Ib=I[bodyIdx],
		     Jb=J[bodyIdx],
		     EIdx  = bodyIdx*12,
		     i, j;

		real Dx = dx[Ib];
	
		// uB = integral (u * delta * dxdy)
		// E = E_hat * R^-1 => divide E_hat by Dx
	
		// populate x-components
		for(j=Jb-1; j<=Jb+1; j++)
		{
			for(i=Ib-2; i<=Ib+1; i++)
			{
				ERows[EIdx] = bodyIdx;
				ECols[EIdx] = j*(nx-1) + i;
				EVals[EIdx] = Dx*delta(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
				EIdx++;
			}
		}

		// populate y-components
		for(j=Jb-2; j<=Jb+1; j++)
		{
			for(i=Ib-1; i<=Ib+1; i++)
			{
				ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
				ECols[EIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
				EVals[EIdx+12*totalPoints-12] = Dx*delta(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
				EIdx++;
			}
		}
	}
}

/**
 * \brief Computes elements of the interpolation matrix.
 *
 * \param ERows row index of elements of the interpolation matrix
 * \param ECols column index of elements of the interpolation matrix
 * \param EVals value of elements of the interpolation matrix
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 * \param x x-component of grid points
 * \param y y-component of grid points
 * \param dx cell-widths in the x-direction
 * \param totalPoints number of body points (all bodies included)
 * \param xB x-coordinate of body points (all bodies included)
 * \param yB y-coordinate of body points (all bodies included)
 * \param I x-index of the cell in which the body point is located
 * \param J y-index of the cell in which the body point is located
 */
__global__ \
void generateE(int *ERows,  int *ECols,  real *EVals,
               int nx, int ny, real *x, real *y, real *dx,
               int totalPoints, real *xB, real *yB, int *I, int *J)
{
	int bodyIdx = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(bodyIdx < totalPoints)
	{
		int  Ib=I[bodyIdx],
			 Jb=J[bodyIdx],
			 EIdx  = bodyIdx*12,
			 i, j;

		real Dx = dx[Ib];
	
		// uB = integral u * delta * dxdy = Ehat * u
		// E = Ehat * R^-1 => divide by Dx
		// E = Dx * delta
	
		// populate x-components
		for(j=Jb-1; j<=Jb+1; j++)
		{
			for(i=Ib-2; i<=Ib+1; i++)
			{
				ERows[EIdx] = bodyIdx;
				ECols[EIdx] = j*(nx-1) + i;
				EVals[EIdx] = Dx*deltaDeviceE(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
				EIdx++;
			}
		}

		// populate y-components
		for(j=Jb-2; j<=Jb+1; j++)
		{
			for(i=Ib-1; i<=Ib+1; i++)
			{
				ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
				ECols[EIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
				EVals[EIdx+12*totalPoints-12] = Dx*deltaDeviceE(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
				EIdx++;
			}
		}
	}
}

} // end of namespace kernels
