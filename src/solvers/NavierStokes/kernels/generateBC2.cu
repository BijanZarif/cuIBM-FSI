#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file generateBC2.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the kernels to generate elements of the right hand-side
 *        of the Poisson solver.
 */


#include "generateBC2.h"


/**
 * \namespace kernels
 * \brief Contains all custom-written CUDA kernels.
 */
namespace kernels
{

/**
 * \brief Computes inhomogeneous terms of the discrete divergence operator 
 *        from the bottom and top boundaries at the v-velocity locations.
 *
 * \param bc2 array that contains boundary conditions
 * \param yminus bottom-boundary velocities
 * \param yplus top-boundary velocities
 * \param dx cell-widths in the x-direction
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 */
__global__
void fillBC2_v(real *bc2, real *yminus, real *yplus, real *dx, int nx, int ny)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=nx)
		return;
	bc2[i] -= yminus[i+nx-1]*dx[i];
	bc2[(ny-1)*nx + i] += yplus[i+nx-1]*dx[i];
}

/**
 * \brief Computes inhomogeneous terms of the discrete divergence operator 
 *        from the left and right boundaries at the u-velocity locations.
 *
 * \param bc2 array that contains boundary conditions
 * \param xminus left-boundary velocities
 * \param xplus right-boundary velocities
 * \param dy cell-widths in the x-direction
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 */
__global__
void fillBC2_u(real *bc2, real *xminus, real *xplus, real *dy, int nx, int ny)
{
	int j = threadIdx.x + blockIdx.x*blockDim.x;
	if(j>=ny)
		return;
	bc2[j*nx] -= xminus[j]*dy[j];
	bc2[j*nx+nx-1] += xplus[j]*dy[j];
}

/**
 * \brief Computes inhomogeneous terms of the discrete divergence operator 
 *        from the no-slip constraint at the body-point locations.
 *
 * \param bc2 array that contains boundary conditions
 * \param uB x-component of the body-velocity
 * \param vB y-component of the body-velcoity
 * \param totalPoints number of body-points (all bodies included)
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 */
__global__
void fillBC2_uvB(real *bc2, real *uB, real *vB, int totalPoints, int nx, int ny)
{
	int k = threadIdx.x + blockIdx.x*blockDim.x;
	if(k>=totalPoints)
		return;
	bc2[nx*ny + k] = uB[k];
	bc2[nx*ny + k + totalPoints] = vB[k];
}

} // end of namespace kernels
