#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file generateQT.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Implementation of the kernels to generate the gradient matrix and interpolation matrix.
 */


#include "generateQT.h"


/**
 * \brief Discrete  delta function defined by Roma et al. (1999).
 *
 * \param x x- or y- component of the vector defined between two points
 * \param h the grid-spacing
 *
 * \return the value of the discrete delta function 
 */
__device__ \
real dhRomaDeviceQT(real x, real h)
{
	real r = fabs(x)/h;
	
	if(r>1.5)
		return 0.0;
	else if(r>0.5 && r<=1.5)
		return 1.0/(6*h)*( 5.0 - 3.0*r - sqrt(-3.0*(1-r)*(1-r) + 1.0) );
	else
		return 1.0/(3*h)*( 1.0 + sqrt(-3.0*r*r + 1.0) );
}

/**
 * \brief Two-dimensional discrete delta function.
 *
 * \param x x-component of the vector defined between two points
 * \param y y-component of the vector defined between two points
 * \param h the grid-spacing
 *
 * \return the value of the discrete delta function in 2D
 */
__device__ \
real deltaDeviceQT(real x, real y, real h)
{
	return dhRomaDeviceQT(x, h) * dhRomaDeviceQT(y, h);
}

/**
 * \namespace kernels
 * \brief Contains all custom-written CUDA kernels.
 */
namespace kernels
{

/**
 * \brief To be documented.
 */
__global__ \
void updateQ(int *QRows, int *QCols, real *QVals, int QSize, int *tags)
{
	int I = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(I < QSize)
	{
		QVals[I] *= (tags[QRows[I]] == -1);
	}
}

__global__ \
void updateQT(int *QTRows, int *QTCols, real *QTVals, int QTSize, int *tags, real *coeffs)
{
	int I = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(I < QTSize)
	{
		int  col = QTCols[I];
		real val = QTVals[I];
		QTCols[I] = (tags[col]==-1)*col + (tags[col]!=-1)*tags[col];
		QTVals[I] = (tags[col]==-1)*val + (tags[col]!=-1)*coeffs[col]*val;
	}
}

/**
 * \brief Generates the divergence matrix.
 *
 * \param QTRows row index of elements of the  divergence matrix
 * \param QTCols column index of elements of the divergence matrix
 * \param QTVals value of elements of the divergence matrix
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 */
void generateQT(int *QTRows, int *QTCols, real *QTVals, int nx, int ny)
{
	int  numU = (nx-1)*ny;
	
	int Iu, Iv;
	int row = 0;
	int num_elements = 0;
	
	/// QT is an (np + 2*nb) x nuv matrix
	
	/// Generate the GT part
	for(int j=0; j<ny; j++)
	{
		for(int i=0; i<nx; i++)
		{
			Iu = j*(nx-1) + i;
			Iv = j*nx + i + numU;
			
			if(i>0)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iu - 1;
				QTVals[num_elements] = 1;
				num_elements++;
			}
			if(i<nx-1)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iu;
				QTVals[num_elements] = -1;
				num_elements++;
			}
			if(j>0)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iv - nx;
				QTVals[num_elements] = 1;
				num_elements++;
			}
			if(j<ny-1)
			{
				QTRows[num_elements] = row;
				QTCols[num_elements] = Iv;
				QTVals[num_elements] = -1;
				num_elements++;
			}
			row++;
		}
	}
}

/**
 * \brief Updates elements of the divergence matrix and the interpolation matrix.
 *
 *
 * \param QTRows row index of elements of the matrix
 * \param QTCols column index of elements of the matrix
 * \param QTVals value of elements of the matrix
 * \param ERows row index of elements of the interpolation matrix
 * \param ECols column index of elements of the interpolation matrix
 * \param EVals value of elements of the interpolation matrix
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 * \param x x-component of grid points
 * \param y y-component of grid points
 * \param dx cell-widths in the x-direction
 * \param totalPoints number of body-points (all bodies included)
 * \param xB x-component of body-points (all bodies included)
 * \param yB y-component of body-points (all bodies included)
 * \param I x-index of grid cells in which body points are located
 * \param J y-index of grid cells in which body points are located
 */
__global__ \
void updateQT(int *QTRows, int *QTCols, real *QTVals,
              int *ERows,  int *ECols,  real *EVals,
              int nx, int ny, real *x, real *y, real *dx,
              int totalPoints, real *xB, real *yB, int *I, int *J)
{
	int bodyIdx = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(bodyIdx >= totalPoints)
		return;
	
	int  Ib=I[bodyIdx],
	     Jb=J[bodyIdx],
	     QTIdx = 4*nx*ny - 2*(nx+ny) + bodyIdx*12,
	     EIdx  = bodyIdx*12,
	     i, j;

	real Dx = dx[Ib];
	
	// populate x-components
	for(j=Jb-1; j<=Jb+1; j++)
	{
		for(i=Ib-2; i<=Ib+1; i++)
		{
			QTRows[QTIdx] = bodyIdx + nx*ny;
			ERows[EIdx] = bodyIdx;
			
			QTCols[QTIdx] = j*(nx-1) + i;
			ECols[EIdx] = QTCols[QTIdx];
			
			QTVals[QTIdx] = Dx*deltaDeviceQT(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
			EVals[EIdx] = QTVals[QTIdx];
			
			QTIdx++;
			EIdx++;
		}
	}

	// populate y-components
	for(j=Jb-2; j<=Jb+1; j++)
	{
		for(i=Ib-1; i<=Ib+1; i++)
		{
			QTRows[QTIdx+12*totalPoints-12] = bodyIdx + nx*ny + totalPoints;
			ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
			
			QTCols[QTIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
			ECols[EIdx+12*totalPoints-12] = QTCols[QTIdx+12*totalPoints-12];
			
			QTVals[QTIdx+12*totalPoints-12] = Dx*deltaDeviceQT(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
			EVals[EIdx+12*totalPoints-12] = QTVals[QTIdx+12*totalPoints-12];
			
			QTIdx++;
			EIdx++;
		}
	}
}

/**
 * \brief Updates the divergence matrix and the interpolation matrix..
 *
 * \param QTRows row index of elements of the matrix
 * \param QTCols column index of elements of the matrix
 * \param QTVals value of elements of the matrix
 * \param ERows row index of elements of the interpolation matrix
 * \param ECols column index of elements of the interpolation matrix
 * \param EVals value of elements of the interpolation matrix
 * \param nx number of cells in the x-direction
 * \param ny number of cells in the y-direction
 * \param x x-component of grid points
 * \param y y-component of grid points
 * \param dx cell-widths in the x-direction
 * \param totalPoints number of body-points (all bodies included)
 * \param xB x-component of body-points (all bodies included)
 * \param yB y-component of body-points (all bodies included)
 * \param I x-index of grid cells in which body points are located
 * \param J y-index of grid cells in which body points are located
 */
void updateQTHost(int *QTRows, int *QTCols, real *QTVals,
              int *ERows,  int *ECols,  real *EVals,
              int nx, int ny, real *x, real *y, real *dx,
              int totalPoints, real *xB, real *yB, int *I, int *J)
{
	for(int bodyIdx=0; bodyIdx<totalPoints; bodyIdx++)
	{
		int  Ib=I[bodyIdx],
		     Jb=J[bodyIdx],
		     QTIdx = 4*nx*ny - 2*(nx+ny) + bodyIdx*12,
		     EIdx  = bodyIdx*12,
		     i, j;

		real Dx = dx[Ib];
	
		// populate x-components
		for(j=Jb-1; j<=Jb+1; j++)
		{
			for(i=Ib-2; i<=Ib+1; i++)
			{
				QTRows[QTIdx] = bodyIdx + nx*ny;
				ERows[EIdx] = bodyIdx;
			
				QTCols[QTIdx] = j*(nx-1) + i;
				ECols[EIdx] = QTCols[QTIdx];
			
				QTVals[QTIdx] = Dx*delta(x[i+1]-xB[bodyIdx], 0.5*(y[j]+y[j+1])-yB[bodyIdx], Dx);
				EVals[EIdx] = QTVals[QTIdx];
			
				QTIdx++;
				EIdx++;
			}
		}

		// populate y-components
		for(j=Jb-2; j<=Jb+1; j++)
		{
			for(i=Ib-1; i<=Ib+1; i++)
			{
				QTRows[QTIdx+12*totalPoints-12] = bodyIdx + nx*ny + totalPoints;
				ERows[EIdx+12*totalPoints-12] = bodyIdx + totalPoints;
			
				QTCols[QTIdx+12*totalPoints-12] = j*nx + i + (nx-1)*ny;
				ECols[EIdx+12*totalPoints-12] = QTCols[QTIdx+12*totalPoints-12];
			
				QTVals[QTIdx+12*totalPoints-12] = Dx*delta(0.5*(x[i]+x[i+1])-xB[bodyIdx], y[j+1]-yB[bodyIdx], Dx);
				EVals[EIdx+12*totalPoints-12] = QTVals[QTIdx+12*totalPoints-12];
			
				QTIdx++;
				EIdx++;
			}
		}
	}
}

} // end of namespace kernels
