#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file parseDomainFile.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \brief Parse the input file \a domain.yaml to obtain information about the 
 *        computational grid.
 */


#include <fstream>
#include <yaml-cpp/yaml.h>
#include "io.h"


/**
 * \namespace io
 * \brief Contains functions related to I/O tasks.
 */
namespace io
{

using std::string;

/**
 * \brief Overloads the operator >>. Gets information from the parsed domain file.
 *
 * \param node the parsed file
 * \param D instance of the class \c domain to be filled
 */
void operator >> (const YAML::Node &node, domain &D)
{
	string dir;
	real start;
	int  numCells;
	
	node["direction"] >> dir;
	node["start"] >> start;
	
	if (dir=="x")
		D.nx = 0;
	else if(dir=="y")
		D.ny = 0;
	
	const YAML::Node &subDomains = node["subDomains"];
	// first pass
	for (unsigned int i=0; i<subDomains.size(); i++)
	{
		subDomains[i]["cells"] >> numCells;
		if (dir=="x")
			D.nx += numCells;
		else if(dir=="y")
			D.ny += numCells;
	}
	
	// allocate memory
	int  beg = 0;
	if(dir=="x")
	{
		D.x.resize(D.nx+1);
		D.dx.resize(D.nx);
		D.xD.resize(D.nx+1);
		D.dxD.resize(D.nx);
		D.x[beg] = start;
	}
	if(dir=="y")
	{
		D.y.resize(D.ny+1);
		D.dy.resize(D.ny);	
		D.yD.resize(D.ny+1);
		D.dyD.resize(D.ny);
		D.y[beg] = start;
	}
	
	// second pass
	real end, stretchRatio, h;
	for (unsigned int i=0; i<subDomains.size(); i++)
	{
		subDomains[i]["end"] >> end;
		subDomains[i]["cells"] >> numCells;
		subDomains[i]["stretchRatio"] >> stretchRatio;
		
		if(fabs(stretchRatio-1.0) < 1.0e-6)
		{
			h = (end - start)/numCells;
			for(int j=beg; j<beg+numCells; j++)
			{
				if(dir=="x")
				{
					D.dx[j]  = h;
					D.x[j+1] = D.x[j] + D.dx[j];
				}
				else if(dir=="y")
				{
					D.dy[j]  = h;
					D.y[j+1] = D.y[j] + D.dy[j];
				}	
			}
		}
		else
		{
			h = (end - start)*(stretchRatio-1)/(pow(stretchRatio, numCells)-1);
			for(int j=beg; j<beg+numCells; j++)
			{
				if(dir=="x")
				{
					D.dx[j]  = h*pow(stretchRatio, j-beg);
					D.x[j+1] = D.x[j] + D.dx[j];
				}
				else if(dir=="y")
				{
					D.dy[j]  = h*pow(stretchRatio, j-beg);
					D.y[j+1] = D.y[j] + D.dy[j];
				}
			}
		}
		beg += numCells;
		start = end;
	}
	
	if(dir=="x")
	{
		D.xD  = D.x;
		D.dxD = D.dx;
	}
	else if(dir=="y")
	{
		D.yD  = D.y;
		D.dyD = D.dy;
	}
}

/**
 * \brief Parses the \a domain file and generates the computational grid.
 *
 * \param domFile the file that contains information about the computational grid
 * \param D instance of the class \c domain that will be filled with information about the computational grid
 */
void parseDomainFile(std::string &domFile, domain &D)
{
	std::ifstream fin(domFile.c_str());
	YAML::Parser  parser(fin);
	YAML::Node    doc;
	parser.GetNextDocument(doc);

	for (unsigned int i=0; i<doc.size(); i++)
		doc[i] >> D;
		
	D.xu.resize(D.nx-1);
	D.yu.resize(D.ny);
	D.xv.resize(D.nx);
	D.yv.resize(D.ny-1);
	
	int i, j;
	for(i=0; i<D.nx-1; i++)
	{
		D.xu[i] = D.x[i+1];
		D.xv[i] = (D.x[i]+D.x[i+1])/2.0;
	}
	D.xv[i] = (D.x[i]+D.x[i+1])/2.0;
	
	for(j=0; j<D.ny-1; j++)
	{
		D.yu[j] = (D.y[j]+D.y[j+1])/2.0;
		D.yv[j] = D.y[j+1];
	}
	D.yu[j] = (D.y[j]+D.y[j+1])/2.0;
}

} // end namespace io
